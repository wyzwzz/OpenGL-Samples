#include "hip/hip_runtime.h"
//
// Created by wyz on 2022/5/16.
//
#define _USE_MATH_DEFINES
#include <cmath>
#include <iostream>
#include "volume_kernel.cuh"
#include "hip/hip_vector_types.h"

#include <hiprand/hiprand_kernel.h>
typedef hiprandStatePhilox4_32_10_t Rand_state;
#define rand(state) hiprand_uniform(state)

__device__ inline bool intersect_volume_box(
    float &tmin, const float3 &raypos, const float3 &raydir)
{
    const float x0 = (-0.5f - raypos.x) / raydir.x;
    const float y0 = (-0.5f - raypos.y) / raydir.y;
    const float z0 = (-0.5f - raypos.z) / raydir.z;
    const float x1 = ( 0.5f - raypos.x) / raydir.x;
    const float y1 = ( 0.5f - raypos.y) / raydir.y;
    const float z1 = ( 0.5f - raypos.z) / raydir.z;

    tmin = fmaxf(fmaxf(fmaxf(fminf(z0,z1), fminf(y0,y1)), fminf(x0,x1)), 0.0f);
    const float tmax = fminf(fminf(fmaxf(z0,z1), fmaxf(y0,y1)), fmaxf(x0,x1));
    return (tmin < tmax);
}
__device__ inline bool in_volume(
    const float3 &pos)
{
    return fmaxf(fabsf(pos.x), fmaxf(fabsf(pos.y), fabsf(pos.z))) < 0.5f;
}

__device__ inline float get_extinction(
    const VolumeKernelParams &kernel_params,
    const float3 &p)
{
        float3 pos = p + make_float3(0.5f, 0.5f, 0.5f);

        float scalar = tex3D<float>(kernel_params.volume_tex,pos.x,pos.y,pos.z);

        float4 value = tex1D<float4>(kernel_params.volume_tf_tex,scalar);
//        printf("scalar: %f, value w: %f\n",scalar,value.w);
        return value.w < 0.001f ? 0.f : kernel_params.max_extinction;

        if(scalar < 0.2f) return 0.f;
        else return kernel_params.max_extinction;
}

__device__ inline bool sample_interaction(
    Rand_state &rand_state,
    float3 &ray_pos,
    const float3 &ray_dir,
    const VolumeKernelParams &kernel_params)
{
    float t = 0.0f;
    float3 pos;
    do {
        t -= logf(max(0.1f,1.0f - rand(&rand_state))) / kernel_params.max_extinction;

        pos = ray_pos + ray_dir * t;
        if (!in_volume(pos))
            return false;

    } while (get_extinction(kernel_params, pos) < rand(&rand_state) * kernel_params.max_extinction);

    ray_pos = pos;
    return true;
}
__device__ inline float3 trace_volume(
    Rand_state &rand_state,
    float3 &ray_pos,
    float3 &ray_dir,
    const VolumeKernelParams &kernel_params){
    float t0;
    float3 w = make_float3(1.f);

    if (intersect_volume_box(t0, ray_pos, ray_dir)){
        ray_pos += ray_dir * t0;
        unsigned int num_interactions = 0;

        while (sample_interaction(rand_state, ray_pos, ray_dir, kernel_params)){
            if (num_interactions++ >= kernel_params.max_interactions)
                return make_float3(0.0f, 0.0f, 0.0f);
            float3 volume_sample_pos = ray_pos + float3{0.5f,0.5f,0.5f};
            float scalar = tex3D<float>(kernel_params.volume_tex,volume_sample_pos.x,volume_sample_pos.y,volume_sample_pos.z);
            float4 value = tex1D<float4>(kernel_params.volume_tf_tex,scalar);
            w *= make_float3(value.x,value.y,value.z);

            float t = max(w.x,max(w.y,w.z));

            if(t < 0.7f && num_interactions > 5){
                float q = max(0.05f,1.f - t);
                if(rand(&rand_state) < q)
                    break ;
                w /= 1.f - q;
            }
            // Sample isotropic phase function.
            const float phi = (float)(2.0 * M_PI) * rand(&rand_state);
            const float cos_theta = 1.0f - 2.0f * rand(&rand_state);
            const float sin_theta = sqrtf(1.0f - cos_theta * cos_theta);
            ray_dir = make_float3(
                cosf(phi) * sin_theta,
                sinf(phi) * sin_theta,
                cos_theta);
        }
    }

    const float4 texval = tex2D<float4>(
        kernel_params.env_tex,
        atan2f(ray_dir.z, ray_dir.x) * (float)(0.5 / M_PI) + 0.5f,
        acosf(fmaxf(fminf(ray_dir.y, 1.0f), -1.0f)) * (float)(1.0 / M_PI));
    return make_float3(texval.x * w.x, texval.y * w.y, texval.z * w.z);
}

__global__ void volume_rt_kernel(VolumeKernelParams kernel_params)
{
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= kernel_params.resolution.x || y >= kernel_params.resolution.y)
        return;

    const unsigned int idx = y * kernel_params.resolution.x + x;

    Rand_state rand_state;
    hiprand_init(idx, 0, kernel_params.iteration * 4096, &rand_state);

    const float inv_res_x = 1.0f / (float)kernel_params.resolution.x;
    const float inv_res_y = 1.0f / (float)kernel_params.resolution.y;
    const float pr = (2.0f * ((float)x + rand(&rand_state)) * inv_res_x - 1.0f);
    const float pu = (2.0f * ((float)y + rand(&rand_state)) * inv_res_y - 1.0f);
    const float aspect = (float)kernel_params.resolution.y * inv_res_x;
    float3 ray_pos = kernel_params.cam_pos;
    float3 ray_dir = normalize(kernel_params.cam_dir * kernel_params.cam_focal + kernel_params.cam_right * pr - kernel_params.cam_up * aspect * pu);


    float3 value = trace_volume(rand_state, ray_pos, ray_dir, kernel_params);
//    value = ray_dir;
//    value = ray_pos;
    if (kernel_params.iteration == 0)
        kernel_params.accum_buffer[idx] = value;
    else
        kernel_params.accum_buffer[idx] = kernel_params.accum_buffer[idx] + (value - kernel_params.accum_buffer[idx]) / (float)(kernel_params.iteration + 1);

    float3 val = kernel_params.accum_buffer[idx] * kernel_params.exposure_scale;
    val.x *= (1.0f + val.x * 0.1f) / (1.0f + val.x);
    val.y *= (1.0f + val.y * 0.1f) / (1.0f + val.y);
    val.z *= (1.0f + val.z * 0.1f) / (1.0f + val.z);
    unsigned int r = (unsigned int)(255.0f * fminf(powf(fmaxf(val.x, 0.0f), (float)(1.0 / 2.2)), 1.0f));
    unsigned int g = (unsigned int)(255.0f * fminf(powf(fmaxf(val.y, 0.0f), (float)(1.0 / 2.2)), 1.0f));
    unsigned int b = (unsigned int)(255.0f * fminf(powf(fmaxf(val.z, 0.0f), (float)(1.0 / 2.2)), 1.0f));

    kernel_params.display_buffer[idx] = 0xff000000 | (r << 16) | (g << 8) | b;
}
